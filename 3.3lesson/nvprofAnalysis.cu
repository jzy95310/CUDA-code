#include "hip/hip_runtime.h"
/*********************************************************************************************
 * file name  : nvprofAnalysis.cu
 * author     : 权 双
 * date       : 2023-08-12
 * brief      : nvprof性能分析
***********************************************************************************************/

/**
 * nvprof是一个可执行文件。在CUDA5.0之后，英伟达才开始使用nvprof作为命令行分析工具，执行命令为：nvprof ./exe_name，
 * 其中exe_name是使用NVCC编译后的可执行文件的名称。
 * nvprof主要用于程序的性能分析，可显示数据传递、内存初始化、核函数计算等操作所占用的时间。
 * 注意：在计算量不大时，数据在主机和设备间进行传递的时间可能远大于核函数计算的时间，在这种情况下使用GPU
 * 进行计算可能并不会带来很大的性能提升。
 */

#include <stdio.h>
#include "../tools/common.cuh"

#define NUM_REPEATS 10

__device__ float add(const float x, const float y)
{
    return x + y;
}

__global__ void addFromGPU(float *A, float *B, float *C, const int N)
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int id = tid + bid * blockDim.x; 
    if (id >= N) return;
    C[id] = add(A[id], B[id]);
    
}


void initialData(float *addr, int elemCount)
{
    for (int i = 0; i < elemCount; i++)
    {
        addr[i] = (float)(rand() & 0xFF) / 10.f;
    }
    return;
}


int main(void)
{
    // 1、设置GPU设备
    setGPU();

    // 2、分配主机内存和设备内存，并初始化
    int iElemCount = 4096;                     // 设置元素数量
    size_t stBytesCount = iElemCount * sizeof(float); // 字节数
    
    // （1）分配主机内存，并初始化
    float *fpHost_A, *fpHost_B, *fpHost_C;
    fpHost_A = (float *)malloc(stBytesCount);
    fpHost_B = (float *)malloc(stBytesCount);
    fpHost_C = (float *)malloc(stBytesCount);
    if (fpHost_A != NULL && fpHost_B != NULL && fpHost_C != NULL)
    {
        memset(fpHost_A, 0, stBytesCount);  // 主机内存初始化为0
        memset(fpHost_B, 0, stBytesCount);
        memset(fpHost_C, 0, stBytesCount);
    
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }


    // （2）分配设备内存，并初始化
    float *fpDevice_A, *fpDevice_B, *fpDevice_C;
    ErrorCheck(hipMalloc((float**)&fpDevice_A, stBytesCount), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((float**)&fpDevice_B, stBytesCount), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((float**)&fpDevice_C, stBytesCount), __FILE__, __LINE__);
    if (fpDevice_A != NULL && fpDevice_B != NULL && fpDevice_C != NULL)
    {
        ErrorCheck(hipMemset(fpDevice_A, 0, stBytesCount), __FILE__, __LINE__); // 设备内存初始化为0
        ErrorCheck(hipMemset(fpDevice_B, 0, stBytesCount), __FILE__, __LINE__);
        ErrorCheck(hipMemset(fpDevice_C, 0, stBytesCount), __FILE__, __LINE__);
    }
    else
    {
        printf("fail to allocate memory\n");
        free(fpHost_A);
        free(fpHost_B);
        free(fpHost_C);
        exit(-1);
    }

    // 3、初始化主机中数据
    srand(666); // 设置随机种子
    initialData(fpHost_A, iElemCount);
    initialData(fpHost_B, iElemCount);
    
    // 4、数据从主机复制到设备
    ErrorCheck(hipMemcpy(fpDevice_A, fpHost_A, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__); 
    ErrorCheck(hipMemcpy(fpDevice_B, fpHost_B, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__);
    ErrorCheck(hipMemcpy(fpDevice_C, fpHost_C, stBytesCount, hipMemcpyHostToDevice), __FILE__, __LINE__);


    // 5、调用核函数在设备中进行计算
    dim3 block(32);
    dim3 grid((iElemCount + block.x - 1) / 32);

    addFromGPU<<<grid, block>>>(fpDevice_A, fpDevice_B, fpDevice_C, iElemCount);    // 调用核函数

    // 6、将计算得到的数据从设备传给主机
    ErrorCheck(hipMemcpy(fpHost_C, fpDevice_C, stBytesCount, hipMemcpyDeviceToHost), __FILE__, __LINE__);

    // 7、释放主机与设备内存
    free(fpHost_A);
    free(fpHost_B);
    free(fpHost_C);
    ErrorCheck(hipFree(fpDevice_A), __FILE__, __LINE__);
    ErrorCheck(hipFree(fpDevice_B), __FILE__, __LINE__);
    ErrorCheck(hipFree(fpDevice_C), __FILE__, __LINE__);

    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);
    return 0;
}

